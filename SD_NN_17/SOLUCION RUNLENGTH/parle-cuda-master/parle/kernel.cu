#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "helper_timer.h"
#include "hip/hip_runtime_api.h"

#include "hemi/grid_stride_range.h"
#include "hemi/launch.h"

#include "chag/pp/prefix.cuh"
#include "chag/pp/reduce.cuh"


#include <conio.h> 

namespace pp = chag::pp;

// Arreglos de memoria global
int* g_symbolsOut;
int* g_countsOut;
int* g_in;
int* g_decompressed;

// Memoria de dispositivo usada en PARLE
int* d_symbolsOut;
int* d_countsOut;
int* d_in;
int* d_totalRuns;
int* d_backwardMask;
int* d_scannedBackwardMask;
int* d_compactedBackwardMask;

const int NUM_TESTS = 11;
const int Tests[NUM_TESTS] = {
	10000, // 10K
	50000, // 50K
	100000, // 100K
	200000, // 200K
	500000, // 500K
	1000000, // 1M
	2000000, // 2M
	5000000, // 5M
	10000000, // 10M
	20000000, // 20M
	30000000, // 30M
};

const int PROFILING_TESTS = 100;
const int MAX_N = 1 << 26; // tama�o m�ximo de cualquier array que usamos.

void parleDevice(int *d_in, int n,
	int* d_symbolsOut,
	int* d_countsOut,
	int* d_totalRuns
	);

int parleHost(int *h_in, int n,
	int* h_symbolsOut,
	int* h_countsOut);

int rleCpu(int *in, int n,
	int* symbolsOut,
	int* countsOut);

__global__ void compactKernel(int* g_in, int* g_scannedBackwardMask, int* g_compactedBackwardMask, int* g_totalRuns, int n) {
	for (int i : hemi::grid_stride_range(0, n)) {

		if (i == (n - 1)) {
			g_compactedBackwardMask[g_scannedBackwardMask[i] + 0] = i + 1;
			*g_totalRuns = g_scannedBackwardMask[i];
		}

		if (i == 0) {
			g_compactedBackwardMask[0] = 0;
		}
		else if (g_scannedBackwardMask[i] != g_scannedBackwardMask[i - 1]) {
			g_compactedBackwardMask[g_scannedBackwardMask[i] - 1] = i;
		}
	}
}

__global__ void scatterKernel(int* g_compactedBackwardMask, int* g_totalRuns, int* g_in, int* g_symbolsOut, int* g_countsOut) {
	int n = *g_totalRuns;

	for (int i : hemi::grid_stride_range(0, n)) {
		int a = g_compactedBackwardMask[i];
		int b = g_compactedBackwardMask[i + 1];

		g_symbolsOut[i] = g_in[a];
		g_countsOut[i] = b - a;
	}
}

__global__ void maskKernel(int *g_in, int* g_backwardMask, int n) {
	for (int i : hemi::grid_stride_range(0, n)) {
		if (i == 0)
			g_backwardMask[i] = 1;
		else {
			g_backwardMask[i] = (g_in[i] != g_in[i - 1]);
		}
	}
}

void PrintArray(int* arr, int n){
	for (int i = 0; i < n; ++i){
		printf("%d, ", arr[i]);
	}
	printf("\n");
}

char errorString[256];

bool verifyCompression(
	int* original, int n,
	int* compressedSymbols, int* compressedCounts, int totalRuns){

	// descomprimir.
	int j = 0;

	int sum = 0;
	for (int i = 0; i < totalRuns; ++i) {
		sum += compressedCounts[i];
	}

	if (sum != n) {
		sprintf(errorString, "El tama�o descomprimido y original no son iguales %d != %d\n", n, sum);

		for (int i = 0; i < totalRuns; ++i){
			int symbol = compressedSymbols[i];
			int count = compressedCounts[i];

			printf("%d, %d\n", count, symbol);
		}
		return false;
	}

	for (int i = 0; i < totalRuns; ++i){
		int symbol = compressedSymbols[i];
		int count = compressedCounts[i];

		for (int k = 0; k < count; ++k){
			g_decompressed[j++] = symbol;
		}
	}

	// Verifica la compresion.
	for (int i = 0; i < n; ++i) {
		if (original[i] != g_decompressed[i]){

			sprintf(errorString, "El arreglo descomprimido y original no son iguales at %d, %d != %d\n", i, original[i], g_decompressed[i]);
			return false;
		}
	}

	return true;
}

// Obteniendo datos de prueba aleatorio para compresion.
// El tipo de datos generados son:
// 1,1,1,1,4,4,4,4,7,7,7,7,....
// de manera que hay muchas secuencias repetidas. 
int* generateCompressibleRandomData(int n){
	int val = rand() % 10;

	for (int i = 0; i < n; ++i) {
		g_in[i] = val;

		if (rand() % 6 == 0){
			val = rand() % 10;
		}
	}
	return g_in;
}


// Obtener datos de prueba aleatorio para compresion.
// El tipo de datos generados son:
// 1,5,8,4,2,6,....
// de manera completamente aleatoria.
int* generateRandomData(int n){
	for (int i = 0; i < n; ++i) {
		g_in[i] = rand() % 10;;

	}
	return g_in;
}

// usamos f para datos comprimidos con RLE y verificar la compression. 
template<typename F>
void unitTest(int* in, int n, F f, bool verbose)
{
	int totalRuns = f(in, n, g_symbolsOut, g_countsOut);

	if (verbose) {
		printf("n = %d\n", n);
		printf("Tama�o original  : %d\n", n);
		printf("Tama�o comprimido: %d\n", totalRuns * 2);
	}

	if (!verifyCompression(
		in, n,
		g_symbolsOut, g_countsOut, totalRuns)) {
		printf("Prueba fallida %s\n", errorString);
		PrintArray(in, n);

		exit(1);
	}
	else {
		if (verbose)
			printf("Prueba correcta!\n\n");
	}
}

// Perfil de implementacion de RLE en CPU
template<typename F, typename G>
void profileCpu(F rle, G dataGen) {
	for (int i = 0; i < NUM_TESTS; ++i) {
		int n = Tests[i];
		int* in = dataGen(n);

		StartCounter();

		for (int i = 0; i < PROFILING_TESTS; ++i) {
			rle(in, n, g_symbolsOut, g_countsOut);
		}
		printf("Para n = %d, en tiempo %.5f microsegundos\n", n, (GetCounter() / ((float)PROFILING_TESTS)) * 1000.0f);
		//printf("%.5f\n", (GetCounter() / ((float)PROFILING_TESTS)) * 1000.0f);

		// Prueba de unidad para asegurarse de que la compresion es correcta.
		unitTest(in, n, rle, false);
	}
}

// Perfil de implementacion de RLE en GPU
template<typename F, typename G>
void profileGpu(F rle, G dataGen) {

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < NUM_TESTS; ++i) {

		int n = Tests[i];
		int* in = dataGen(n);

		// Tranfiere los datos al dispositivo.
		CUDA_CHECK(hipMemcpy(d_in, in, n*sizeof(int), hipMemcpyHostToDevice));

		// graba.
		hipEventRecord(start);
		for (int i = 0; i < PROFILING_TESTS; ++i) {
			parleDevice(d_in, n, d_symbolsOut, d_countsOut, d_totalRuns);
		}
		hipEventRecord(stop);
		hipDeviceSynchronize();

		// Prueba de unidad para asegurarse de que la compresi�n es correcta
		unitTest(in, n, rle, false);

		float ms;
		hipEventElapsedTime(&ms, start, stop);



		printf("Para n = %d, en tiempo %.5f microsegundos\n", n, (ms / ((float)PROFILING_TESTS)) *1000.0f);
		//printf("%.5f\n", (ms / ((float)PROFILING_TESTS)) *1000.0f);
	}
}

// Ejecuta varios test en la implemetacion(f) de RLE.
template<typename F>
void runTests(int a, F f) {
	printf("EMPIEZAN PRUEBAS UNITARIAS\n");

	for (int i = 4; i < a; ++i) {
		for (int k = 0; k < 30; ++k) {
			int n = 2 << i;

			if (k != 0) {
				// en la primera prueba se hacen con valores buenos para 'n'
				// en las otras dos pruebas se hacen con valores ligeramente ramdomizados
				n = (int)(n * (0.6f + 1.3f * (rand() / (float)RAND_MAX)));
			}

			int* in = generateCompressibleRandomData(n);

			unitTest(in, n, f, true);
		}
		printf("-------------------------------\n\n");
	}
}

int main(){

	srand(1000);
	CUDA_CHECK(hipSetDevice(0));

	// Asignar recursos al dispositivo, estas matrices se utilizan globalmente en el programa
	CUDA_CHECK(hipMalloc((void**)&d_backwardMask, MAX_N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_scannedBackwardMask, MAX_N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_compactedBackwardMask, (MAX_N + 1) * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_in, MAX_N* sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_countsOut, MAX_N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_symbolsOut, MAX_N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_totalRuns, sizeof(int)));

	// asignar recursos al servidor. 
	g_in = new int[MAX_N];
	g_decompressed = new int[MAX_N];
	g_symbolsOut = new int[MAX_N];
	g_countsOut = new int[MAX_N];

	// Ejecutamos este codigo para correr muchas pruebas unitarias.
	/*
	runTests(21, rleCpu);
	runTests(21, parleHost);
	*/

	// Ejecutamos este codigo para perfilar el rendimiento. 
	printf("Perfil: datos aleatorios en CPU\n");
	profileCpu(rleCpu, generateRandomData);

	printf("Perfil: datos comprimibles en CPU\n");
	profileCpu(rleCpu, generateCompressibleRandomData);


	printf("Perfil: datos aleatorios en GPU\n");
	profileGpu(parleHost, generateRandomData);

	printf("Perfil: datos comprimibles GPU\n");
	profileGpu(parleHost, generateCompressibleRandomData);



	// Ejecutamos este c�digo cuando queremos ejecutar NVPP en el algoritmo.
	/*
	int n = 1 << 23;
	unitTest(generateCompressibleRandomData(1<<23), n, rleGPU, true);
	*/

	// liberamos los arreglos del dispositivo.
	CUDA_CHECK(hipFree(d_backwardMask));
	CUDA_CHECK(hipFree(d_scannedBackwardMask));
	CUDA_CHECK(hipFree(d_compactedBackwardMask));
	CUDA_CHECK(hipFree(d_in));
	CUDA_CHECK(hipFree(d_countsOut));
	CUDA_CHECK(hipFree(d_symbolsOut));
	CUDA_CHECK(hipFree(d_totalRuns));

	CUDA_CHECK(hipDeviceReset());

	// liberamos los arreglos del servidor.
	delete[] g_in;
	delete[] g_decompressed;

	delete[] g_symbolsOut;
	delete[] g_countsOut;

	while (getch() != '\n');


	return 0;
}



// implementation of RLE on CPU.
int rleCpu(int *in, int n, int* symbolsOut, int* countsOut){

	if (n == 0)
		return 0; // nada para comprimir!

	int outIndex = 0;
	int symbol = in[0];
	int count = 1;

	for (int i = 1; i < n; ++i) {
		if (in[i] != symbol) {
			// ha terminado la ejecucion.
			// ejecuta la salida.
			symbolsOut[outIndex] = symbol;
			countsOut[outIndex] = count;
			outIndex++;

			// y empieza una nueva ejecucion:
			symbol = in[i];
			count = 1;
		}
		else {
			++count; // aun no termina la ejecucion.
		}
	}

	// ultima ejecucion de salida. 
	symbolsOut[outIndex] = symbol;
	countsOut[outIndex] = count;
	outIndex++;

	return outIndex;
}
// En el CPU se hace la preparacion para ejecutar parle, lanza PARLE en GPU y tranfiere los resultados de la CPU
int parleHost(int *h_in, int n,
	int* h_symbolsOut,
	int* h_countsOut){

	int h_totalRuns;

	// tranfiere datos de entrada al dispositivo.
	CUDA_CHECK(hipMemcpy(d_in, h_in, n*sizeof(int), hipMemcpyHostToDevice));

	// ejecuta.    
	parleDevice(d_in, n, d_symbolsOut, d_countsOut, d_totalRuns);

	// transfiere los resultados del dispositivo al host.
	CUDA_CHECK(hipMemcpy(h_symbolsOut, d_symbolsOut, n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_countsOut, d_countsOut, n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&h_totalRuns, d_totalRuns, sizeof(int), hipMemcpyDeviceToHost));

	return h_totalRuns;
}

void scan(int* d_in, int* d_out, int N) {
	pp::prefix_inclusive(d_in, d_in + N, d_out);
}

// ejecuta parle en la GPU
void parleDevice(int *d_in, int n,
	int* d_symbolsOut,
	int* d_countsOut,
	int* d_totalRuns
	){
	hemi::hipLaunchByPtr(maskKernel, d_in, d_backwardMask, n);
	scan(d_backwardMask, d_scannedBackwardMask, n);
	hemi::hipLaunchByPtr(compactKernel, d_in, d_scannedBackwardMask, d_compactedBackwardMask, d_totalRuns, n);
	hemi::hipLaunchByPtr(scatterKernel, d_compactedBackwardMask, d_totalRuns, d_in, d_symbolsOut, d_countsOut);
}


